/*
  ============================================================================
  Name        : GameOfLife.cu
  Author      : José Carlos
  Version     :
  Copyright   : 
  Description : CUDA compute reciprocals
  ============================================================================
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define N 100
#define DEAD_CELL 0
#define ALIVE_CELL 1

void printCurrentState(int ** __state) {
  system("clear");
  for(int i = 0; i < N;i++)
    {
      for(int j = 0; j < N;j++) {
	if(__state[i][j] == DEAD_CELL)
	  printf(" ");
	else
	  printf("+");
      }
      printf("\n");
    }
}


__device__ int MOVEMENTS_SIZE = 8;

__device__ int MOVEMENTS[8][2] =
  {
    {0, 1},
    {1, 0},
    {0, -1},
    {-1, 0},
    {1, 1},
    {-1, -1},
    {1, -1},
    {-1, 1}
  };

__global__ void kernel_game_of_life(int * status, int * new_status) {
  int i, j, neighbour_x, neighbour_y, neighbours_a, neighbours_d, neighbours_status, current_status;

  i = (threadIdx.x + blockIdx.x * blockDim.x) % N;
  j = (threadIdx.x + blockIdx.x * blockDim.x) / N;

  current_status = status[i*N+j];

  neighbours_a = neighbours_d = 0;
  for(int z = 0; z < MOVEMENTS_SIZE;z++) {
    neighbour_x = i + MOVEMENTS[z][0];
    neighbour_y = j + MOVEMENTS[z][1];

    if(neighbour_x < 0 || neighbour_y < 0 || neighbour_x >= N || neighbour_y >= N) {
      continue;
    }
    neighbours_status = status[neighbour_x*N + neighbour_y];

    if(neighbours_status == 0) {
      neighbours_d++;
    } else if(neighbours_status == 1) {
      neighbours_a++;
    }
  }


  if(current_status == 0 && neighbours_a == 3) {
    new_status[i*N+j] = 1;
  } else if (current_status == 1 && (neighbours_a > 3 || neighbours_a < 2)) {
    new_status[i*N+j] = 0;
  }
}


int * matrix2vector(int ** matrix, int size) {
  int * vector = (int*)malloc(size*size*sizeof(int)), i, j;
  for (i = 0; i < size; i++) {
    for (j = 0; j < size; j++) {
      vector[i*size + j] = matrix[i][j];
    }
  }
  return vector;
}

void vector2matrix(int * vector, int ** matrix, int size) {
  int i, j;
  for (i = 0; i < size; i++) {
    for (j = 0; j < size; j++) {
      matrix[i][j] = vector[i*size + j];
    }
  }
}
int main(void)
{
  int i, j, minGridSize, blockSize, gridSize;

  int ** __status_matrix, * __status_vector, * __status_dev_vector, * __new_status_dev_vector;

  __status_matrix = (int**)malloc(sizeof(int*)*N);

  for(i = 0; i < N;i++)
    __status_matrix[i] = (int*)malloc(sizeof(int)*N);

  srand(time(NULL));

  for(i = 0; i < N;i++)
    {
      for(j = 0; j < N;j++)
    	{
	  __status_matrix[i][j] = rand() % 2;
    	}
    }

  __status_vector = matrix2vector(__status_matrix, N);


  hipMalloc((void**)&__status_dev_vector, N*N*sizeof(int));
  hipMalloc((void**)&__new_status_dev_vector, N*N*sizeof(int));

  hipMemcpy(__status_dev_vector, __status_vector, N*N* sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(__new_status_dev_vector, __status_vector, N*N* sizeof(int), hipMemcpyHostToDevice);

  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel_game_of_life, 0, N*N);
  gridSize = (N*N + blockSize - 1) / blockSize;

  printCurrentState(__status_matrix);

  while(true) {
    kernel_game_of_life<<<gridSize, blockSize >>>(__status_dev_vector, __new_status_dev_vector);
    hipDeviceSynchronize();

    hipMemcpy(__status_vector, __new_status_dev_vector, N*N* sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(__status_dev_vector, __new_status_dev_vector, N*N* sizeof(int), hipMemcpyDeviceToDevice);

    vector2matrix(__status_vector, __status_matrix, N);
    printCurrentState(__status_matrix);
    usleep(5000);
  }


}
